#include "hip/hip_runtime.h"
#pragma once

/*
    Structure of Includes:
    - System/Language
    - Vendor/External Libs
    - Same Directory/Self Files
*/

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>

#include "GL/glew.h"
#include "GLFW/glfw3.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtx/norm.hpp"
#include "glm/gtc/type_ptr.hpp"
#include "glm/gtc/vec1.hpp"
#include "glm/gtc/constants.hpp"
#include "imgui/imconfig.h"
#include "imgui/imgui.h"
#include "imgui/imgui_impl_glfw.h"
#include "imgui/imgui_impl_opengl3.h"

#include "Renderer.h"
#include "Shader.h"
#include "ParticleSystem.cuh"

#define GImGui MyImGuiTLS

// FIX: Structure of Arrays - Array of Structs - Memory access for threads better ?

// Update Particle Function - Will run on the GPU
__global__ void UpdateParticle(ParticleSystem::Particle* new_particles, float timestep)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (new_particles[i].life_rem <= 0.0f)
    {
        new_particles[i].alive = false;
    }
    new_particles[i].life_rem -= timestep;
    new_particles[i].vel.y = new_particles[i].vel.y + new_particles[i].acc * timestep;
    new_particles[i].pos += new_particles[i].pos + new_particles[i].vel * timestep;
}

int main()
{
    GLFWwindow* window;

    /* Initialize the library */
    glfwSetErrorCallback(GLFW_error);
    if (!glfwInit())
        return -1;

    // GL 3.2 + GLSL 150
    const char* glsl_version = "#version 150";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    //glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    //glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    //glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(640, 480, "Particle Demo Scene", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);

    //Checks to make sure glew is initialized
    if (glewInit() != GLEW_OK)
        std::cout << "Error!" << std::endl;

    //Gets version of OpenGL used
    std::cout << glGetString(GL_VERSION) << std::endl;

    //Defining my MVP matricies
    //Setting everything to normalized device coordinates for ease of numbers used
    glm::mat4 proj = glm::ortho(-3.0f , 3.0f, -3.0f, 3.0f, -1.0f, 1.0f);
    //Not chaning the view matrix
    glm::mat4 view = glm::translate(glm::mat4(1.0f), glm::vec3(0, 0, 0));
    //Settting the model in the middle of the screen
    glm::mat4 model = glm::translate(glm::mat4(1.0f), glm::vec3(0, 0, 0));
    glm::mat4 mvp = proj * view * model;

    //Geometry of the Particles
    //Vertex Array
    float vertices[] = 
    {
        -0.15f, -0.25f, 0.0f,
         0.15f, -0.25f, 0.0f,
         0.15f,  0.25f, 0.0f,
        -0.15f,  0.25f, 0.0f
    };

    //Index Array
    unsigned int indices[] =
    {
        0, 1, 2,
        2, 3, 0
    };
    
    //Buffer setup
    //Creating and binding the VAO
    unsigned int vao;
    GLCall(glCreateVertexArrays(1, &vao));
    GLCall(glBindVertexArray(vao));

    //Creating and binding the VBO
    unsigned int geomVB, geomIB;
    GLCall(glCreateBuffers(1, &geomVB));
    GLCall(glBindBuffer(GL_ARRAY_BUFFER, geomVB));
    GLCall(glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW));

    //Defining what the vertices consist of
    GLCall(glEnableVertexArrayAttrib(geomVB, 0));
    GLCall(glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), 0));

    //Creating and binding the IBO
    GLCall(glCreateBuffers(1, &geomIB));
    GLCall(glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, geomIB));
    GLCall(glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW));

    //Setup for Renderer
    Renderer renderer;
    renderer.EnableBlend();

    //Setup for Particle System
    unsigned int num_particles = 10000;
    ParticleSystem particlesystem(num_particles);
    int particles_per_frame = 1;
    particlesystem.setParticleShader("res/shaders/particle.shader");
    
    // CUDA program setup - Goal: 1 thread per particle
    int blocks = num_particles/100;
    int threads_per_block = particlesystem.getContainerSize() / blocks;

    // Using hipMalloc and hipMemcpy
    int pool_byte_size = particlesystem.getContainerSize() * sizeof(ParticleSystem::Particle) + 1;
    //hipDeviceSetLimit(hipLimitMallocHeapSize, pool_byte_size);
    // Pointer to Vector particle container
    ParticleSystem::Particle* cpu_bin = &particlesystem.particle_container[0];

    // Initialize gpu container
    ParticleSystem::Particle* gpu_bin;
    hipMalloc((void**) &gpu_bin, pool_byte_size);

    // Using thrust library
    //thrust::device_vector<ParticleSystem::Particle> gpu_particle_container = particlesystem.getParticleContainer();
    //thrust::device_vector<ParticleSystem::Particle> gpu_particle_container(particlesystem.getParticleContainer().begin(), particlesystem.getParticleContainer().end());

    //Initializing particle properties
    ParticleProperties particleproperties;
    particleproperties.col_beg = { 254 / 255.0f, 212 / 255.0f, 123 / 255.0f, 1.0f };
    particleproperties.col_end = { 254 / 255.0f, 109 / 255.0f, 41 / 255.0f, 1.0f };
    particleproperties.size_beg = 0.25f;
    particleproperties.size_var = 0.05f;
    particleproperties.size_end = 0.0f;
    particleproperties.life = 10.0f;
    particleproperties.vel = { 0.0f, 0.7f };
    particleproperties.vel_var = { .1f, .05f };
    particleproperties.pos = { 0.0f, 0.0f };
    particleproperties.acc = -0.05f;

    // Setup Dear ImGui Context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    //ImGui::SetCurrentContext(MyImGuiTLS);  // ???????
    //ImGuiIO& io = ImGui::GetIO(); (void)io;
    //io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
    //io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls

    // Setup Platform/Renderer backends
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);
    
    // Setup Dear ImGui style
    ImGui::StyleColorsDark(); 

    // Initializes times
    float time = (float)glfwGetTime();
    float lastframetime = 0;

    // Loop until the user closes the window
    while (!glfwWindowShouldClose(window))
    {
        //Render here

        //Operations to receieve my deltatime
        time = (float)glfwGetTime();
        float timestep = time - lastframetime;
        lastframetime = time;

        //Calls glClear(GL_COLOR_BUFFER_BIT)
        renderer.Clear();
        
        // Dear ImGui Call New Frame 
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();
        
        /// Particle System Loops ///

        // Emit Particle(s) Function (per frame)
        for (int i = 0; i < particles_per_frame; i++)
            particlesystem.Emit(particleproperties);

        //particlesystem.Update(timestep);

        /// CUDA ///
        
        // Multithreaded Physics Update Function 
        hipMemcpy(gpu_bin, cpu_bin, pool_byte_size, hipMemcpyHostToDevice);
        UpdateParticle<<<blocks,threads_per_block>>>(gpu_bin, timestep);
        hipDeviceSynchronize();
        hipMemcpy(cpu_bin, gpu_bin, pool_byte_size, hipMemcpyDeviceToHost);

        // CUDA Error Checking
        hipError_t error = hipPeekAtLastError(); // check if kernel launch worked
        if (error == 0) error = hipDeviceSynchronize(); // wait for GPU threads to finish
        if (error != 0) printf("CUDA Error %s (%d)", hipGetErrorString(error), (int)error);
        
        // Render Update Function
        particlesystem.Render(mvp, vao);
        
        // Dear ImGui Active Components
        ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
        ImGui::DragFloat("Life Time", &particleproperties.life, 0.1f, 0.0f, 1000.0f);
        ImGui::DragInt("Particles Per Frame", &particles_per_frame, 1, 0, 50);

        // Dear ImGui Render Loop
        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        
        // Swap front and back buffers
        glfwSwapBuffers(window);
        // Poll for and process events
        glfwPollEvents();
    }

    // hipMalloc cleanup
    hipFree(gpu_bin);

    //Night-Night time functions
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    glfwTerminate();
    return 0;
}